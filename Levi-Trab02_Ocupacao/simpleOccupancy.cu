#include "hip/hip_runtime.h"
/**
 * Copyright 1993-2014 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

#include <iostream>
#include <hip/hip_runtime_api.h>         // helper functions for CUDA error check

////////////////////////////////////////////////////////////////////////////////
// Test kernel
//
// This kernel sums each array element. Each thread addresses
// himself with threadIdx and blockIdx, so that it can handle any
// execution configuration, including anything the launch configurator
// API suggests.
////////////////////////////////////////////////////////////////////////////////
__global__ void myKernel(int *array, int arrayCount)
{
    extern __shared__ int dynamicSmem[];
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    
    if (idx < arrayCount) {
        array[idx] += array[idx];
    }
}

////////////////////////////////////////////////////////////////////////////////
// Potential occupancy calculator
//
// The potential occupancy is calculated according to the kernel and
// execution configuration the user desires. Occupancy is defined in
// terms of active blocks per multiprocessor, and the user can convert
// it to other metrics.
//
// This wrapper routine computes the occupancy of kernel, and reports
// it in terms of active warps / maximum warps per SM.
////////////////////////////////////////////////////////////////////////////////
static double reportPotentialOccupancy(void *kernel, int blockSize, size_t dynamicSMem)
{
    int device;
    hipDeviceProp_t prop;

    int numBlocks;
    int activeWarps;
    int maxWarps;

    double occupancy;

    checkCudaErrors(hipGetDevice(&device));
    checkCudaErrors(hipGetDeviceProperties(&prop, device));

    checkCudaErrors(hipOccupancyMaxActiveBlocksPerMultiprocessor(
                        &numBlocks,
                        kernel,
                        blockSize,
                        dynamicSMem));

    activeWarps = numBlocks * blockSize / prop.warpSize;
    maxWarps = prop.maxThreadsPerMultiProcessor / prop.warpSize;

    occupancy = (double)activeWarps / maxWarps;

    return occupancy;
}

////////////////////////////////////////////////////////////////////////////////
// Occupancy-based launch configurator
//
// The launch configurator, hipOccupancyMaxPotentialBlockSize and
// hipOccupancyMaxPotentialBlockSizeVariableSMem, suggests a block
// size that achieves the best theoretical occupancy. It also returns
// the minimum number of blocks needed to achieve the occupancy on the
// whole device.
//
// This launch configurator is purely occupancy-based. It doesn't
// translate directly to performance, but the suggestion should
// nevertheless be a good starting point for further optimizations.
//
// This function configures the launch based on the "automatic"
// argument, records the runtime, and reports occupancy and runtime.
////////////////////////////////////////////////////////////////////////////////
static int launchConfig(int *array, int arrayCount, int blockSize)
{
    int minGridSize;
    int gridSize;
	unsigned int bytes = arrayCount * sizeof(int);

    size_t dynamicSMemUsage = 0;

    hipEvent_t start;
    hipEvent_t end;

    float elapsedTime;
    
    double potentialOccupancy;

    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&end));

    if (blockSize == 0) {
        checkCudaErrors(hipOccupancyMaxPotentialBlockSize(
                            &minGridSize,
                            &blockSize,
                            (void*)myKernel,
                            dynamicSMemUsage,
                            arrayCount));

        std::cout << "Suggested block size: " << blockSize
                  << ". Minimum grid size for maximum occupancy: " << minGridSize << std::endl;
    }

    // Round up
    //
    gridSize = (arrayCount + blockSize - 1) / blockSize;
    
	std::cout << "GridSize: " << gridSize << ". Block size: " << blockSize << ". dynamicSMemUsage: " << dynamicSMemUsage << ". arrayCount: " << arrayCount << ". bytes: " << bytes;

    // Launch and profile
    //
    checkCudaErrors(hipEventRecord(start));
    myKernel<<<gridSize, blockSize, dynamicSMemUsage>>>(array, arrayCount);
    checkCudaErrors(hipEventRecord(end));

    checkCudaErrors(hipDeviceSynchronize());

    // Calculate occupancy
    //
    potentialOccupancy = reportPotentialOccupancy((void*)myKernel, blockSize, dynamicSMemUsage);

    std::cout << ". Potential occupancy: " << potentialOccupancy * 100 << "%";

    // Report elapsed time
    //
    checkCudaErrors(hipEventElapsedTime(&elapsedTime, start, end));
    std::cout << ". Elapsed time: " << elapsedTime << "ms" << std::endl;
    
    return 0;
}

////////////////////////////////////////////////////////////////////////////////
// The test
////////////////////////////////////////////////////////////////////////////////
static int test(int blockSize, int count)
{
    int *array;
    int *dArray;
    int size = count * sizeof(int);

    array = new int[count];

    for (int i = 0; i < count; i += 1) {
        array[i] = i;
    }

    checkCudaErrors(hipMalloc(&dArray, size));
    checkCudaErrors(hipMemcpy(dArray, array, size, hipMemcpyHostToDevice));

    for (int i = 0; i < count; i += 1) {
        array[i] = 0;
    }

    launchConfig(dArray, count, blockSize);

    checkCudaErrors(hipMemcpy(array, dArray, size, hipMemcpyDeviceToHost));
    checkCudaErrors(hipFree(dArray));

    // Verify the return data
    //
    for (int i = 0; i < count; i += 1) {
        if (array[i] != i + i) {
            std::cout << "element " << i << " expected " << i + i << " actual " << array[i] << std::endl;
            return 1;
        }
    }

    checkCudaErrors(hipDeviceReset());

    delete[] array;

    return 0;
}

////////////////////////////////////////////////////////////////////////////////
// Main
//
// The sample runs the test with automatically configured launch and
// manually configured launch, and reports the occupancy and
// performance.
////////////////////////////////////////////////////////////////////////////////
int main()
{
    int status;
	for (int j = 0; j <= 4; ++j) {
		int count = 100000 * pow(2, j);
		std::cout << "Occupancy with array size: " << count << std::endl << std::endl;

		std::cout << "[ Automatic, occupancy-based configuration ]" << std::endl;
		status = test(0, count);
		if (status) {
			std::cerr << "Test failed\n" << std::endl;
			return -1;
		}        
		std::cout << std::endl;

		for (int i = 8; i <= 512; i*= 2) {
//			std::cout << "[ Manual configuration with " << i
	//				  << " threads per block ]" << std::endl;

			status = test(i, count);
			if (status) {
				std::cerr << "Test failed\n" << std::endl;
	//			return -1;
			}

			std::cout << std::endl;
		}
	}

    std::cout << std::endl;
    std::cout << "Test PASSED\n" << std::endl;
    
    return 0;
}
